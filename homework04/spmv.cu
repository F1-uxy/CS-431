#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

#include "spmv.h"

template <class T>
__global__ void
spmv_kernel_ell(unsigned int* col_ind, T* vals, int m, int n, int nnz, 
                    double* x, double* b)
{
    
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m) 
    {
        T sum = 0.0;

        for (int i = 0; i < n; i++) 
        {
            int idx = row * n + i;

            unsigned int col = col_ind[idx];
            T val = vals[idx];

            if (col < nnz && val != -1) 
            {
                sum += val * x[col];
            }
        }

        b[row] = sum;
    }
   
    /*
    unsigned int tid = threadIdx.x;
    unsigned int row = blockIdx.x;
    unsigned int i = tid;

    extern __shared__ T Local[];

    Local[tid] = 0;

    while(i < n)
    {
        int idx = row * n + i;

        unsigned int col = col_ind[idx];
        T val = vals[idx];

        if(val != -1 && col < nnz)
        {
            Local[tid] += val * x[col];
        }

        i += blockDim.x;
    }

    __syncthreads();

    for(int i = blockDim.x/2; i > 0; i >>= 1)
    {
        if(tid < i)
        {
            Local[tid] += Local[tid + i];
        }
    
        __syncthreads();
    }


    if(tid == 0 && row < m)
    {
        b[blockIdx.x] = Local[0];
    }*/
}


void spmv_gpu_ell(unsigned int* col_ind, double* vals, int m, int n, int nnz, 
                  double* x, double* b, int nthreads)
{
    // timers
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;

    // GPU execution parameters
    unsigned int blocks = m; 
    unsigned int threads = nthreads; 
    unsigned int shared = threads * sizeof(double);

    dim3 dimGrid(blocks, 1, 1);
    dim3 dimBlock(threads, 1, 1);

    checkCudaErrors(hipEventRecord(start, 0));
    for(unsigned int i = 0; i < MAX_ITER; i++) {
        hipDeviceSynchronize();
        spmv_kernel_ell<double><<<dimGrid, dimBlock, shared>>>(col_ind, vals, 
                                                               m, n, nnz, x, b);
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("  Exec time (per itr): %0.8f s\n", (elapsedTime / 1e3 / MAX_ITER));
}




void allocate_ell_gpu(unsigned int* col_ind, double* vals, int m, int n, 
                      int nnz, double* x, unsigned int** dev_col_ind, 
                      double** dev_vals, double** dev_x, double** dev_b)
{
    // copy ELL data to GPU and allocate memory for output
    // COMPLETE THIS FUNCTION
    

    CopyData<unsigned int>(col_ind, m * n, sizeof(unsigned int), dev_col_ind);
    CopyData<double>(vals, m * n, sizeof(double), dev_vals);
    CopyData<double>(x, m, sizeof(double), dev_x);

    checkCudaErrors(hipMalloc((void**)dev_b, m * sizeof(double)));  
}

void allocate_csr_gpu(unsigned int* row_ptr, unsigned int* col_ind, 
                      double* vals, int m, int n, int nnz, double* x, 
                      unsigned int** dev_row_ptr, unsigned int** dev_col_ind,
                      double** dev_vals, double** dev_x, double** dev_b)
{
    // copy CSR data to GPU and allocate memory for output
    // COMPLETE THIS FUNCTION

    CopyData<unsigned int>(row_ptr, m + 1, sizeof(unsigned int), dev_row_ptr);
    CopyData<unsigned int>(col_ind, nnz, sizeof(unsigned int), dev_col_ind);
    CopyData<double>(vals, nnz, sizeof(double), dev_vals);
    CopyData<double>(x, m, sizeof(double), dev_x);

    checkCudaErrors(hipMalloc((void**)dev_b, m * sizeof(double)));
}

void get_result_gpu(double* dev_b, double* b, int m)
{
    // timers
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;


    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipMemcpy(b, dev_b, sizeof(double) * m, 
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("  Pinned Host to Device bandwidth (GB/s): %f\n",
         (m * sizeof(double)) * 1e-6 / elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

template <class T>
void CopyData(
  T* input,
  unsigned int N,
  unsigned int dsize,
  T** d_in)
{
  // timers
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float elapsedTime;

  // Allocate pinned memory on host (for faster HtoD copy)
  T* h_in_pinned = NULL;
  checkCudaErrors(hipHostMalloc((void**) &h_in_pinned, N * dsize));
  assert(h_in_pinned);
  memcpy(h_in_pinned, input, N * dsize);

  // copy data
  checkCudaErrors(hipMalloc((void**) d_in, N * dsize));
  checkCudaErrors(hipEventRecord(start, 0));
  checkCudaErrors(hipMemcpy(*d_in, h_in_pinned,
                             N * dsize, hipMemcpyHostToDevice));
  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
  printf("  Pinned Device to Host bandwidth (GB/s): %f\n",
         (N * dsize) * 1e-6 / elapsedTime);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}


template <class T>
__global__ void
spmv_kernel(unsigned int* row_ptr, unsigned int* col_ind, T* vals, 
            int m, int n, int nnz, double* x, double* b)
{
    // COMPLETE THIS FUNCTION
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m) 
    {
        T sum = 0.0;

        for (unsigned int j = row_ptr[row]; j < row_ptr[row + 1]; j++) 
        {
            unsigned int col = col_ind[j];
            T val = vals[j];
            sum += val * x[col];
        }

        b[row] = sum;
    }
}


void spmv_gpu(unsigned int* row_ptr, unsigned int* col_ind, double* vals,
              int m, int n, int nnz, double* x, double* b, int nthreads)
{
    // timers
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;

    // GPU execution parameters
    // 1 thread block per row
    // 64 threads working on the non-zeros on the same row
    unsigned int blocks = m; 
    unsigned int threads = nthreads; 
    unsigned int shared = threads * sizeof(double);

    dim3 dimGrid(blocks, 1, 1);
    dim3 dimBlock(threads, 1, 1);

    checkCudaErrors(hipEventRecord(start, 0));
    for(unsigned int i = 0; i < MAX_ITER; i++) {
        hipDeviceSynchronize();
        spmv_kernel<double><<<dimGrid, dimBlock, shared>>>(row_ptr, col_ind, 
                                                           vals, m, n, nnz, 
                                                           x, b);
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("  Exec time (per itr): %0.8f s\n", (elapsedTime / 1e3 / MAX_ITER));

}
