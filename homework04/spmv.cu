#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

#include "spmv.h"

template <class T>
__global__ void
spmv_kernel_ell(unsigned int* col_ind, T* vals, int m, int n, int nnz, 
                double* x, double* b)
{

    // COMPLETE THIS FUNCTION
}



void spmv_gpu_ell(unsigned int* col_ind, double* vals, int m, int n, int nnz, 
                  double* x, double* b)
{
    // timers
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;

    // GPU execution parameters
    unsigned int blocks = m; 
    unsigned int threads = 64; 
    unsigned int shared = threads * sizeof(double);

    dim3 dimGrid(blocks, 1, 1);
    dim3 dimBlock(threads, 1, 1);

    checkCudaErrors(hipEventRecord(start, 0));
    for(unsigned int i = 0; i < MAX_ITER; i++) {
        hipDeviceSynchronize();
        spmv_kernel_ell<double><<<dimGrid, dimBlock, shared>>>(col_ind, vals, 
                                                               m, n, nnz, x, b);
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("  Exec time (per itr): %0.8f s\n", (elapsedTime / 1e3 / MAX_ITER));

}




void allocate_ell_gpu(unsigned int* col_ind, double* vals, int m, int n, 
                      int nnz, double* x, unsigned int** dev_col_ind, 
                      double** dev_vals, double** dev_x, double** dev_b)
{
    // copy ELL data to GPU and allocate memory for output
    // COMPLETE THIS FUNCTION
}

void allocate_csr_gpu(unsigned int* row_ptr, unsigned int* col_ind, 
                      double* vals, int m, int n, int nnz, double* x, 
                      unsigned int** dev_row_ptr, unsigned int** dev_col_ind,
                      double** dev_vals, double** dev_x, double** dev_b)
{
    // copy CSR data to GPU and allocate memory for output
    // COMPLETE THIS FUNCTION
}

void get_result_gpu(double* dev_b, double* b, int m)
{
    // timers
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;


    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipMemcpy(b, dev_b, sizeof(double) * m, 
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("  Pinned Host to Device bandwidth (GB/s): %f\n",
         (m * sizeof(double)) * 1e-6 / elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

template <class T>
void CopyData(
  T* input,
  unsigned int N,
  unsigned int dsize,
  T** d_in)
{
  // timers
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float elapsedTime;

  // Allocate pinned memory on host (for faster HtoD copy)
  T* h_in_pinned = NULL;
  checkCudaErrors(hipHostMalloc((void**) &h_in_pinned, N * dsize));
  assert(h_in_pinned);
  memcpy(h_in_pinned, input, N * dsize);

  // copy data
  checkCudaErrors(hipMalloc((void**) d_in, N * dsize));
  checkCudaErrors(hipEventRecord(start, 0));
  checkCudaErrors(hipMemcpy(*d_in, h_in_pinned,
                             N * dsize, hipMemcpyHostToDevice));
  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
  printf("  Pinned Device to Host bandwidth (GB/s): %f\n",
         (N * dsize) * 1e-6 / elapsedTime);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}


template <class T>
__global__ void
spmv_kernel(unsigned int* row_ptr, unsigned int* col_ind, T* vals, 
            int m, int n, int nnz, double* x, double* b)
{
    // COMPLETE THIS FUNCTION
}


void spmv_gpu(unsigned int* row_ptr, unsigned int* col_ind, double* vals,
              int m, int n, int nnz, double* x, double* b)
{
    // timers
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;

    // GPU execution parameters
    // 1 thread block per row
    // 64 threads working on the non-zeros on the same row
    unsigned int blocks = m; 
    unsigned int threads = 64; 
    unsigned int shared = threads * sizeof(double);

    dim3 dimGrid(blocks, 1, 1);
    dim3 dimBlock(threads, 1, 1);

    checkCudaErrors(hipEventRecord(start, 0));
    for(unsigned int i = 0; i < MAX_ITER; i++) {
        hipDeviceSynchronize();
        spmv_kernel<double><<<dimGrid, dimBlock, shared>>>(row_ptr, col_ind, 
                                                           vals, m, n, nnz, 
                                                           x, b);
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("  Exec time (per itr): %0.8f s\n", (elapsedTime / 1e3 / MAX_ITER));

}
